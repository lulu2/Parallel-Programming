#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void mykernel(){
	printf("Hello world from device!\n");
} /* end kernel */

int main(void) 
{
	mykernel<<<1,10>>>();
 	hipDeviceSynchronize();
	printf("Hello World from Host\n");
	return 0;
} /* end main */
