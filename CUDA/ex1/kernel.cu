#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void simpleKernel( int *a )
{

/* calculate my global index in the array */

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

/* assign value to the output array */
/* change code here to change the output */

//	a[idx] = 7;
//	a[idx] = blockIdx.x;
	a[idx] = threadIdx.x;
} /* end simpleKernel */ 


int main()
{
	int dimx = 16;
	int numbytes = dimx * sizeof( int );

/* declare the device and host pointers */

	int *d_a = 0, *h_a = 0; // device and host pointers

/* allocate the memory on host and device */

	h_a = (int *) malloc( numbytes );
	hipMalloc( (void **) &d_a, numbytes );

	if( 0 == h_a || 0 == d_a )
	{
		printf("Couldn't allocate memory!\n");
		return 911;
	} /* end if */

/* initialize GPU memory to 0 */

	hipMemset( d_a, 0, numbytes );

/* setup GPU grid and block */

	dim3 mygrid, myblock;

	myblock.x = 4;
	mygrid.x = dimx / myblock.x;

/* launch the kernel */

	simpleKernel<<< mygrid, myblock >>>( d_a );

/* copy result back to GPU */

	hipMemcpy( h_a, d_a, numbytes, hipMemcpyDeviceToHost );

/* check GPU and CPU data to ensure they are equal */

	for( int i = 0; i < dimx; i++ )
	{
		printf("%d ", h_a[i] );
	} /* end for */
	printf("\n");

/* free the memory and cleanup */

	free( h_a );
	hipFree( d_a );

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 911;
    }

    return 0;
}

