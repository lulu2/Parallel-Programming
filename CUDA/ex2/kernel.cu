#include "hip/hip_runtime.h"

#include <stdio.h>

#define CUDA_ERROR() printf("cuda error is %s\n",hipGetErrorString( hipGetLastError() ));

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	/* enter code here to malloc d_b and d_c */
//        FIXME
        hipMalloc( (void **) &d_b, size );
        hipMalloc( (void **) &d_c, size );
	
        /* setup initial values */

	a = 2;
	b = 7;
	c = -99;

	/* copy inputs to device */

	hipMemcpy( d_a, &a, size, hipMemcpyHostToDevice );
	/* enter code here to copy d_b to device */
        //FIXME
       hipMemcpy( d_b, &b, size, hipMemcpyHostToDevice );
       hipMemcpy( d_c, &c, size, hipMemcpyHostToDevice );
	/* launch the kernel on the GPU */
	/* enter code here */
       // FIXME
        add<<< 4,10 >>>( d_a, d_b, d_c );
	/* copy result back to host */

	hipMemcpy( &c, d_c, size, hipMemcpyDeviceToHost );

	printf("value of c after kernel is %d\n",c);

	/* clean up */

	hipFree( d_a );
	/* enter code here to cudaFree the d_b and d_c pointers */
	hipFree(d_b);
        hipFree(d_c);
	return 0;
} /* end main */
