#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
	/* insert correct index so that each element is calculated by a different thread */
        int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

/* experiment with different values of N */
/* how large can you make it? */
#define N 512

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */

	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );
	hipMemset( d_c, 0, size );

	/* launch the kernel on the GPU */
	/* insert the correct launch parameters to use 1 block and N threads */
	/* how large can you make N?  */
	add<<< 1,N   >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	for( int i = 0; i < N; i++ )
	{
		printf("c[%d] = %d\n",i,c[i]);
	} /* end for */

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
