#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* finish this code to calculate c element-wise from a and b where each block calculates one element */
        int i = blockIdx.x;
	c[i] = a[i] + b[i];
}


/* experiment with different values of N.  */
/* how large can it be? */
#define N 2048*32

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */
	
	hipMalloc( (void **) &d_a, size );
	/* insert code here for d_b and d_c */
        
        hipMalloc( (void **) &d_b, size );
        hipMalloc( (void **) &d_c, size );
	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	/* intializing a, b, c on host */
	
	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	/* insert code to copy b to the device */
        
        hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );
        hipMemset( d_c, 0, size );

	/* launch the kernel on the GPU */
	/* finish this kernel launch with N blocks and 1 thread per block */
	add<<< N, 1  >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

/*	for( int i = 0; i < N; i++ )
	{
		printf(" c[%d] = %d\n",i,c[i]);
	}*/ /* end for */

       printf("The last element is c[%d] = %d\n",N-1,c[N-1]);
	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
