#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockDim.x*blockIdx.x+threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */

	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	add<<< N/512, 512 >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n",0,c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
