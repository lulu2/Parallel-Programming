
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

/* definitions of threadblock size in X and Y directions */

#define THREAD_X 16
#define THREAD_Y 16

/* definition of matrix linear dimension */

#define SIZE 1024

/* macro to index a 1D memory array with 2D indices in column-major order */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* CUDA kernel for naive matrix transpose */

__global__ void naive_cuda_transpose( const int m, double const * const a, double *c )
{
	/* insert code to calculate global row and column of the matrix */
	const int myRow = blockIdx.x * blockDim.x + threadIdx.x;
	const int myCol = blockIdx.y * blockDim.y + threadIdx.y;
	/*const char k=blockDim.x;*/
	/*const char kk=blockIdx.x;*/
	/*printf(k,kk);*/
	if( myRow < m && myCol < m )
	{
		/* insert the indices for accessing the A and C matrices to execute the transpose */
               


		 c[INDX(myRow, myCol, m)] = a[INDX(myCol, myRow, m)];
	} /* end if */
	return;
} /* end naive_cuda_transpose */

void host_transpose( const int m, double const * const a, double *c )
{
	
/* 
 *  naive matrix transpose on CPU goes here.
 */
 
 for( int j = 0; j < m; j++ )
	{
		for( int i = 0; i < m; i++ )
		{
		    c[INDX(i,j,m)] = a[INDX(j,i,m)];
		} /* end for i */
	} /* end for j */

} /* end host_dgemm */

int main( int argc, char *argv[] )
{

    int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

/* declaring pointers for array */

    double *h_a, *h_c;
    double *d_a, *d_c;
 
    size_t numbytes = (size_t) size * (size_t) size * sizeof( double );

/* allocating host memory */

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc h_a\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc h_c\n");
      return 911;
    }

/* allocating device memory */

    hipMalloc( (void**) &d_a, numbytes );
    hipMalloc( (void**) &d_c, numbytes );

/* set result matrices to zero */

    memset( h_c, 0, numbytes );
    hipMemset( d_c, 0, numbytes );

    fprintf( stdout, "Total memory required per matrix is %lf MB\n", 
       (double) numbytes / 1000000.0 );

/* initialize input matrix with random value */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

/* copy input matrix from host to device */

    hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );

/* create and start timer */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

/* call naive cpu transpose function */

    host_transpose( size, h_a, h_c );

/* stop CPU timer */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

/* print CPU timing information */

    fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GB/s\n", 
      8.0 * 2.0 * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );

/* setup threadblock size and grid sizes */

    dim3 threads( THREAD_X, THREAD_Y, 1 );

	/* insert code for proper grids in X and Y directions */
    dim3 blocks( SIZE/THREAD_X, SIZE/THREAD_Y, 1 );

/* start timers */
    hipEventRecord( start, 0 );

/* call naive GPU transpose kernel */

    naive_cuda_transpose<<< blocks, threads >>>( size, d_a, d_c );

/* stop the timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

/* print GPU timing information */

    fprintf(stdout, "Total time GPU is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GB/s\n", 
      8.0 * 2.0 * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );

/* copy data from device to host */

    hipMemset( h_a, 0, numbytes );
    hipMemcpy( h_a, d_c, numbytes, hipMemcpyDeviceToHost );

/* compare GPU to CPU for correctness */

	for( int j = 0; j < size; j++ )
	{
		for( int i = 0; i < size; i++ )
		{
		    if( h_c[INDX(i,j,size)] != h_a[INDX(i,j,size)] ) 
                    {
                      printf("Error in element %d,%d\n", i,j );
                      printf("Host %f, device %d\n",h_c[INDX(i,j,size)],
                                                    h_a[INDX(i,j,size)]);
                    }
		} /* end for i */
	} /* end for j */

/* free the memory */

    free( h_a );
    free( h_c );
    hipFree( d_a );
    hipFree( h_a );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
