#include "hip/hip_runtime.h"
#include <stdio.h>

/* macro to index a 1D memory array with 2D indices in column-major order */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* linear size of the matrices */

#define SIZE 1024


/* CPU matrix multiply function */

void host_dgemm( const int m, const int n, const int k, double const * const a, double const * const b, double *c )
{
	
/* 
 *  naive matrix multiplication loops go here.  triply nested for loop
 *  C = A * B where A and B are matrices
 *  C(i,j) = SUM( A(i,k) * B(k,j), over the index "k", where 0 <= k < (SIZE-1) )   
 */     


/* insert code here */

 for( int j = 0; j < n; j++ )
	{
		for( int i = 0; i < m; i++ )
		{
			for( int koff = 0; koff < k; koff++ )
			{
				/* insert proper index calculations here */
				c[INDX(i,j, m)] += a[INDX( i,koff, m )] * b[INDX( koff,j, n )];
			} /* end for koff */
		} /* end for i */
	} /* end for j */

} /* end host_dgemm */

int main( int argc, char *argv[] )
{

    int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

/* declare host pointers */

    double *h_a, *h_b, *h_cdef;
 
    size_t numbytes = size * size * sizeof( double );

/* allocate host pointers */

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_cdef = (double *) malloc( numbytes );
    if( h_cdef == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

/* set C to zero */

    memset( h_cdef, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

/* initialize A and B on the host */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

/* start timers */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );

/* call host dgemm */

    host_dgemm( size, size, size, h_a, h_b, h_cdef );

/* stop the timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

/* print the results */

    fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );

/* cleanup */

    free( h_a );
    free( h_b );
    free( h_cdef );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
