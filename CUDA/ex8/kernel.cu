#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024
#define THREAD_X 16
#define THREAD_Y 16

/* naive GPU kernel where each element of C is computed by a single thread */

__global__ void GPU_naive( const int m, double const * const a, double const * const b, double * const c )
{

/* insert code to determine my threads's row and col indices in the global C matrix */
	const int myrow = blockDim.x * blockIdx.x + threadIdx.x;
	const int mycol = blockDim.y * blockIdx.y + threadIdx.y;

/* if my row and col are in the C matrix, then calculate that value of C */

	if( myrow < m && mycol < m )
	{
		register double temp = 0.0;
		for( int k = 0; k < m; k++ ) 
		{
			/* insert correct index code here */
			temp += a[INDX( myrow,k, m )] * b[INDX( k,mycol, m )];
		} /* end for */
		/* insert index code to write the output to the C matrix */
		c[INDX( myrow, mycol, m )] = temp;
	} /* end if */

	return;
} /* end GPU_naive */

int main( int argc, char *argv[] )
{

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_c1;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = (size_t ) size * (size_t ) size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	h_c1 = (double *) malloc( numbytes );
    if( h_c1 == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	/* zero out the host memory for C matrices */

    memset( h_c, 0, numbytes );
    memset( h_c1, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

	/* initialize the A and B matrices */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

	/* allocate a, b, c in gpu memory */

    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );
	
	/* copy a and b to device */

	hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    double alpha = 1.0;
    double beta  = 0.0;

	/* start timers */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

	/* call CUBLAS dgemm */

hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print GPU CUBLAS timing information */

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
	/* copy C from device to host for error checking */

    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

	/* reset C on device to zero */

	hipMemset( d_c, 0, numbytes );

	/* setup grid and block sizes */

	dim3 threads( THREAD_X, THREAD_Y, 1 );
	dim3 blocks( size / THREAD_X + 1, size / THREAD_Y + 1, 1 );

	/* start timers */

	hipEventRecord( start, 0 );

	/* call GPU_naive */

	GPU_naive<<< blocks, threads >>> ( size, d_a, d_b, d_c );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print data for GPU naive */

    fprintf(stdout, "Total time GPU NAIVE is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
	/* copy C back to host */
	
	hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost );

    hipblasDestroy( handle );
    hipEventDestroy( start );
    hipEventDestroy( stop );

	/* check CUBLAS versus GPU NAIVE numerical results */

	double temp = 0.0;

	for( int i = 0; i < size * size; i++ )
	{
		temp += ( h_c[i] - h_c1[i] ) * ( h_c[i] - h_c1[i] );
	} /* end for */

	printf("error is %f\n",temp);
	if( temp > 10 ) printf("Error value is suspiciously high!\n");

	/* cleanup */

    hipFree( d_a );
    hipFree( d_b );
	hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_c1 );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
