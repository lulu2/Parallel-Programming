#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>

#define pi_f  3.14159265358979f 

 struct prg
{
   float a, b;

    __host__ __device__
    prg(float _a=0.f, float _b=1.f) : a(_a), b(_b) {};

    __host__ __device__
        float operator()(const unsigned int n) const
        {
            thrust::default_random_engine rng;
            thrust::uniform_real_distribution<float> dist(a, b);
            rng.discard(n);
            return dist(rng);
        }
};


// want to integrate f = sin(x)
struct integrand_functor
{
    __host__ __device__
    float operator()(float x) const
    {
        return sin(x);
    }
};


int main(void)
{
    const int N = 20000000;
    // generate uniform r.v. from a = 0, b = pi
    const float a = 0.0f;
    const float b = pi_f;
    thrust::device_vector<float> numbers(N);
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
   
    thrust::transform(index_sequence_begin,
            index_sequence_begin + N,
            numbers.begin(),
            prg(a,b));

    // evaluate function values at each random numbers
   thrust::device_vector<float> eva(N);
   thrust::transform(numbers.begin(), numbers.end(),eva.begin(), 
               integrand_functor());
    
    float sum = thrust::reduce(eva.begin(), eva.end(), 0.f,
              thrust::plus<float>());
    /*for(int i = 0; i < N; i++)
    {
        std::cout << numbers[i] << std::endl;
	std::cout << eva[i] << std::endl;

    }
    std::cout << sum << std::endl;*/
    
    std::cout << "The integral of sin(x) from " << a << " to " <<b<< " is "<< sum*(b-a)/N << std::endl;
        return 0;
}
