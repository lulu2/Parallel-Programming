#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>

#include "ReadImageData.h"
#include "PCA.h"

__global__ void kmeans_initialize(float *points, float *cores, int num_cluster, int dim, int *d_cluster_master, int* d_change)
{    
     int index = blockIdx.x * blockDim.x + threadIdx.x ;
     float bar=FLT_MAX;
     float distance;
     d_change[index]=0;
     for (int i = 0; i<num_cluster; i++){
        distance=0.0;
        for (int j=0;j<dim;j++){
             distance +=(cores[i*dim+j]-points[index*dim+j])*(cores[i*dim+j]-points[index*dim+j]);   
         }
       if (distance<bar) {
         bar = distance;
         d_cluster_master[index] = i;
            }
     }
     __syncthreads();
}

__global__ void kmeans(float *points, float *cores, int num_cluster, int dim, int *d_cluster_master, int* d_change)
{    
     int index = blockIdx.x * blockDim.x + threadIdx.x ;
     float bar=FLT_MAX;
     float distance;
     d_change[index]=0;
     int currents=d_cluster_master[index];
     for (int i = 0; i<num_cluster; i++){
        distance=0.0;
        for (int j=0;j<dim;j++){
             distance +=(cores[i*dim+j]-points[index*dim+j])*(cores[i*dim+j]-points[index*dim+j]);   
         }
       if (distance<bar) {
         bar = distance;
         d_cluster_master[index] = i;
            }
     }  
       if (d_cluster_master[index]!= currents) {
         d_change[index] = 1;
         }
     __syncthreads();
}

__global__ void updata_cluster(float *points, float *cores, int num_data, int dim, int *d_cluster_master, int *d_number_member, float *sum, int offset)
{   
    int index = threadIdx.x ;
    index+=offset*8;
    d_number_member[index]=0;
    for (int k=0;k<dim;k++){
        sum[index*dim+k]=0.0;
    }
 for (int i = 0; i<num_data; i++){
        if (d_cluster_master[i]==index){
        for (int j=0;j<dim;j++){
            sum[index*dim+j] += points[i*dim+j];
        }
            d_number_member[index]++;
        }
    }

    for (int p=0; p<dim; p++){
        cores[index*dim+p]=sum[index*dim+p]/d_number_member[index];
    }
    __syncthreads();
}

__global__ void data_transform(float *d_input_data, float *d_transform_data, int dim)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x ;
    d_transform_data[index]=d_input_data[(index%dim)*dim+index/dim];
    __syncthreads();
}

int main(int argc, char **argv)
{
    int dim;
    int num_data;
    int num_cluster;
    char* path;


    if (argc != 2) {
        printf("Please input file!\n");
        return 1;
    }
    dim = 500;
    num_cluster = 10;
    path = argv[1];
    int D = 0;


    float* X = readFile(path, &num_data, &D);
    float* input_data = (float*)malloc(num_data*dim*sizeof(float));
    float* P = (float*)malloc(D*dim*sizeof(float));

    PCA(X, num_data, D, dim, P, input_data);

    free(X);
    free(P);

    int size_data = num_data * dim * sizeof(float);
    int size_cluster = num_cluster * dim *sizeof(float);

    float *h_cores=(float*) malloc (size_cluster);
    int *h_change=(int *) malloc (num_data*sizeof(int));
    int *h_cluster_master=(int *) malloc (num_data*sizeof(int));
    int *h_label=(int *) malloc (num_data*sizeof(int));

    int *d_cluster_master;
    float *d_points;
    float *d_cores;
    int *d_change;
    int *d_number_member;
    float *d_sum;
    hipMalloc(&d_sum, size_cluster);
    hipMalloc(&d_cluster_master, num_data*sizeof(int));
    hipMalloc(&d_points, size_data);
    hipMalloc(&d_cores, size_cluster);
    hipMalloc(&d_change, num_data*sizeof(int));
    hipMalloc(&d_number_member, num_cluster*sizeof(int));

    float *h_input_data=input_data;
    float *h_transform_data=(float*) malloc (size_data);
    float *d_input_data;
    float *d_transform_data;
    hipMalloc(&d_input_data, size_data);
    hipMalloc(&d_transform_data, size_data);


//Set labels
    int small_group=num_data/10;
    int big_group=(num_data/10)+1;
    int num_big=num_data%10;
    int num_small=10-num_big;
    
    for (int i=0;i<num_small;i++){
    for (int j=0;j<small_group;j++){
    h_label[i*small_group+j]=i;
    }
    }

    for (int i=num_small;i<10;i++){
    for (int j=0;j<big_group;j++){
    h_label[num_small*small_group+(i-num_small)*big_group+j]=i;
    }
    }

/*
    for (int i=0;i<4;i++){
    for (int j=0;j<409;j++){
    h_label[i*409+j]=i;
    }
    }

    for (int i=4;i<10;i++){
    for (int j=0;j<410;j++){
    h_label[4*409+(i-4)*410+j]=i;
    }
    }
*/

    std::clock_t t1, t2, t3, t4;

    t1 = std::clock();

    hipMemcpy(d_input_data, h_input_data, size_data, hipMemcpyHostToDevice);
    if (num_data<512){
    data_transform<<<1, 256>>>(d_input_data, d_transform_data, dim);
    }
    else {
    data_transform<<<num_data/512, 512>>>(d_input_data, d_transform_data, dim);
    }
    //data_transform<<<8, 512>>>(d_input_data, d_transform_data, dim);
    hipMemcpy(h_transform_data, d_transform_data, size_data, hipMemcpyDeviceToHost);
    t2 = std::clock();


    for (int i=0;i<num_cluster;i++){
        //int tempt = rand() % num_data;
        //std::cout<<tempt<<std::endl;
        int tempt=(i+1)*(num_data/10)-8;
        for (int j=0;j<dim;j++){
            h_cores[i*dim+j]=h_transform_data[tempt*dim+j];
        }
    }

    //std::cout<<num_data<<std::endl;
    //for (int i=0;i<50;i++){
    //std::cout<<h_cores[i]<<std::endl;
    //}

    hipMemcpy(d_points, h_transform_data, size_data, hipMemcpyHostToDevice);
    hipMemcpy(d_cores, h_cores, size_cluster, hipMemcpyHostToDevice);

    t3 = std::clock();
    //Initialize update
    if (num_data<512){
    kmeans_initialize<<<1, 256>>>(d_points, d_cores, num_cluster, dim, d_cluster_master, d_change);
    }
    else {
    kmeans_initialize<<<num_data/512, 512>>>(d_points, d_cores, num_cluster, dim, d_cluster_master, d_change);
    }
    hipDeviceSynchronize();
    updata_cluster<<< 1, 8>>>(d_points, d_cores, num_data, dim, d_cluster_master, d_number_member,d_sum, 0);
    hipDeviceSynchronize();
    updata_cluster<<< 1, 2>>>(d_points, d_cores, num_data, dim, d_cluster_master, d_number_member,d_sum, 1);
    hipDeviceSynchronize();

    //update
    //while(fluctuation>threshold){
    for (int kk=0;kk<50;kk++){
    if (num_data<512){
    kmeans<<<1, 256>>>(d_points, d_cores, num_cluster, dim, d_cluster_master, d_change);
    }
    else {
    kmeans<<<num_data/512, 512>>>(d_points, d_cores, num_cluster, dim, d_cluster_master, d_change);
    }
    hipDeviceSynchronize();
    updata_cluster<<< 1, 8>>>(d_points, d_cores, num_data, dim, d_cluster_master, d_number_member,d_sum, 0);
    hipDeviceSynchronize();
    updata_cluster<<< 1, 2>>>(d_points, d_cores, num_data, dim, d_cluster_master, d_number_member,d_sum, 1);
    hipMemcpy(h_change, d_change, num_data*sizeof(int), hipMemcpyDeviceToHost);
//    fluctuation=0;
 //       for (int m=0;m<num_data;m++){
 //           if (h_change[m]==1){
  //          fluctuation++;
   //         }
     //   }
    }

    int* h_number_member = (int *) malloc (num_cluster*sizeof(int));
    hipMemcpy(h_number_member, d_number_member, num_cluster*sizeof(int), hipMemcpyDeviceToHost);
    t4 = std::clock();

    float time_data_transform=(t2-t1)/(float) CLOCKS_PER_SEC;
    float time_kmeans_transform=(t4-t3)/(float) CLOCKS_PER_SEC;

    std::cout<<"Call for transform_data kernel takes "<<time_data_transform<<" seconds"<<std::endl;
    std::cout<<"Call for k-means kernel takes "<<time_kmeans_transform<<" seconds"<<std::endl;
    
    free(h_change);
    free(h_cores);
    free(h_input_data);
    free(h_transform_data);
    free(h_cluster_master);
    free(h_number_member);
    hipFree(d_points);
    hipFree(d_cores);
    hipFree(d_cluster_master);
    hipFree(d_number_member);
    hipFree(d_change);
    hipFree(d_sum);
    hipFree(d_transform_data);

    return 0;
}

